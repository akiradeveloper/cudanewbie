
#include <hip/hip_runtime.h>
#include <iostream>
typedef float4 Real3;

void puts(Real3 v){
  std::cout << "(" << v.x << " " << v.y << " " << v.z << ")" << std::endl;
}

Real3 operator+(Real3 v1, Real3 v2){
  Real3 p;
  p.x = v1.x + v2.x;
  p.y = v1.y + v2.y;
  p.z = v1.z = v2.z;
  return p;
}

Real3 genReal3(float x, float y, float z){
  Real3 p;
  p.x = x;
  p.y = y;
  p.z = z;
  return p; 
}

int main(void){
  Real3 p = genReal3(10,20,30);

  puts(p);

  Real3 q = genReal3(40,50,60);
  
  puts(p+q);
  
  return 0;
}
  
